#include <cfloat>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
// #include <limits>
#include <random>
#include <utility>
#include <vector>

// Node structure for A* algorithm
struct Node {
  int x, y; // Coordinates
  float gCost, hCost, fCost;
  Node *parent;

  // Constructor
  Node()
      : x(0), y(0), gCost(FLT_MAX), hCost(FLT_MAX), fCost(FLT_MAX),
        parent(nullptr) {}
};

__device__ float manhattanDistance(int x1, int y1, int x2, int y2) {
  return abs(x1 - x2) + abs(y1 - y2);
}

const int patternSize = 10;
const int numPatterns = 4;
int patterns[numPatterns][patternSize][patternSize] = {
    // pattern 1
    {{0, 0, 0, -1, 0, 0, -1, 0, 0, 0},
     {0, -1, 0, 0, -1, 0, 0, -1, 0, 0},
     {0, 0, -1, 0, 0, -1, 0, 0, -1, 0},
     {0, 0, 0, -1, 0, 0, -1, 0, 0, -1},
     {0, -1, 0, 0, -1, 0, 0, -1, 0, 0},
     {0, 0, -1, 0, 0, -1, 0, 0, -1, 0},
     {0, 0, 0, -1, 0, 0, -1, 0, 0, -1},
     {0, -1, 0, 0, -1, 0, 0, -1, 0, 0},
     {0, 0, -1, 0, 0, -1, 0, 0, -1, 0},
     {0, 0, 0, -1, 0, 0, -1, 0, 0, -1}},
    // pattern 2
    {{0, 0, 0, -1, 0, 0, -1, 0, 0, 0},
     {0, 0, -1, 0, 0, -1, 0, 0, -1, 0},
     {0, -1, 0, 0, -1, 0, 0, -1, 0, 0},
     {-1, 0, 0, -1, 0, 0, -1, 0, 0, 0},
     {0, 0, -1, 0, 0, -1, 0, 0, -1, 0},
     {0, -1, 0, 0, -1, 0, 0, -1, 0, 0},
     {-1, 0, 0, -1, 0, 0, -1, 0, 0, 0},
     {0, 0, -1, 0, 0, -1, 0, 0, -1, 0},
     {0, -1, 0, 0, -1, 0, 0, -1, 0, 0},
     {-1, 0, 0, -1, 0, 0, -1, 0, 0, 0}},
    // pattern 3
    {{0, 0, 0, -1, 0, 0, 0, -1, 0, 0},
     {0, -1, 0, -1, 0, -1, 0, -1, 0, -1},
     {0, -1, 0, -1, 0, -1, 0, -1, 0, -1},
     {0, -1, 0, -1, 0, -1, 0, -1, 0, -1},
     {0, -1, 0, -1, 0, -1, 0, -1, 0, -1},
     {0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
     {0, -1, 0, -1, 0, -1, 0, -1, 0, -1},
     {0, -1, 0, -1, 0, -1, 0, -1, 0, -1},
     {0, -1, 0, -1, 0, -1, 0, -1, 0, -1},
     {0, -1, 0, 0, 0, -1, 0, 0, 0, -1}},
    // pattern 4
    {{0, -1, -1, -1, -1, 0, -1, -1, -1, -1},
     {0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
     {-1, -1, -1, -1, -1, 0, -1, -1, -1, 0},
     {0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
     {0, -1, -1, -1, -1, 0, -1, -1, -1, -1},
     {0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
     {-1, -1, -1, -1, -1, 0, -1, -1, -1, 0},
     {0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
     {0, -1, -1, -1, -1, 0, -1, -1, -1, -1},
     {0, 0, 0, 0, 0, 0, 0, 0, 0, 0}},
};

class Environment {
public:
  // Environment(int rows, int cols) : rows(rows), cols(cols) {
  //   map.resize(rows, std::vector<int>(cols, 0));
  //   createWithPatterns();
  // }

  Environment(int rows, int cols) : rows(rows), cols(cols) {
    map.resize(rows, std::vector<int8_t>(cols, 0));
    nodeGrid.resize(rows, std::vector<Node>(cols));
    createWithPatterns();
    initializeNodes();
  }

  void createWithPatterns() {
    for (int i = 0; i < rows; i += patternSize) {
      for (int j = 0; j < cols; j += patternSize) {
        // Choose a pattern and place it in the map
        int patternIndex = std::rand() % numPatterns; // Random pattern index
        for (int pi = 0; pi < patternSize && i + pi < rows; ++pi) {
          for (int pj = 0; pj < patternSize && j + pj < cols; ++pj) {
            map[i + pi][j + pj] = patterns[patternIndex][pi][pj];
          }
        }
      }
    }
  }

  void saveToFile(const std::string &filename) {
    std::ofstream file(filename);
    if (!file.is_open()) {
      std::cerr << "Failed to open file for writing." << std::endl;
      return;
    }

    for (const auto &row : map) {
      for (size_t j = 0; j < row.size(); ++j) {
        file << row[j];
        if (j < row.size() - 1)
          file << ",";
      }
      file << "\n";
    }
  }

  void positionRobotAndPackage() {
    std::vector<std::pair<int, int>> freeSpaces;
    for (int i = 0; i < rows; ++i) {
      for (int j = 0; j < cols; ++j) {
        if (map[i][j] == 0) {
          freeSpaces.emplace_back(i, j);
        }
      }
    }

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> distrib(0, freeSpaces.size() - 1);

    // Randomly select two different positions for the robot and the package
    int index1 = distrib(gen);
    int index2 = distrib(gen);
    while (index1 == index2) {
      index2 = distrib(gen); // Ensure different positions
    }

    robotPos = freeSpaces[index1];
    packagePos = freeSpaces[index2];
  }

  void printRobotAndPackagePosition() {
    std::cout << "Robot Position: (" << robotPos.first << ", "
              << robotPos.second << ")\n";
    std::cout << "Package Position: (" << packagePos.first << ", "
              << packagePos.second << ")\n";
  }

  void initializeNodes() {
    for (int i = 0; i < rows; i++) {
      for (int j = 0; j < cols; j++) {
        nodeGrid[i][j].x = i;
        nodeGrid[i][j].y = j;
      }
    }
  }

private:
  int rows, cols;
  std::vector<std::vector<Node>> nodeGrid;
  std::vector<std::vector<int8_t>> map;
  std::pair<int, int> robotPos;
  std::pair<int, int> packagePos;
};

int main() {
  std::srand(static_cast<unsigned int>(time(nullptr)));

  Environment env(500, 500);
  env.saveToFile("../../wharehouse_ex/500_500.csv");

  env.positionRobotAndPackage();
  env.printRobotAndPackagePosition();

  return 0;
}