#include <cfloat>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <fstream>
#include <iostream>
// #include <limits>
#include <random>
#include <utility>
#include <vector>

// Node structure for A* algorithm
struct Node {
  int x, y; // Coordinates
  float gCost, hCost, fCost;
  Node *parent;

  // Constructor
  Node()
      : x(0), y(0), gCost(FLT_MAX), hCost(FLT_MAX), fCost(FLT_MAX),
        parent(nullptr) {}
};

__device__ float manhattanDistance(int x1, int y1, int x2, int y2) {
  return abs(x1 - x2) + abs(y1 - y2);
}

__global__ void initializeNodesKernel(Node *d_nodes, int rows, int cols) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;

  if (idx < cols && idy < rows) {
    int index = idy * cols + idx;
    d_nodes[index].x = idx;
    d_nodes[index].y = idy;
    d_nodes[index].gCost = FLT_MAX;
    d_nodes[index].hCost = FLT_MAX;
    d_nodes[index].fCost = FLT_MAX;
    d_nodes[index].parent = nullptr;
  }
}

__global__ void aStarKernel(Node *d_nodes, int rows, int cols, int targetX,
                            int targetY) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;

  if (idx < cols && idy < rows) {
    int index = idy * cols + idx;
    d_nodes[index].x = idx;
    d_nodes[index].y = idy;
    d_nodes[index].gCost = FLT_MAX;
    d_nodes[index].hCost = manhattanDistance(idx, idy, targetX, targetY);
    d_nodes[index].fCost = FLT_MAX;
    d_nodes[index].parent = nullptr;
  }
}

const int patternSize = 10;
const int numPatterns = 4;
int patterns[numPatterns][patternSize][patternSize] = {
    // pattern 1
    {{0, 0, 0, -1, 0, 0, -1, 0, 0, 0},
     {0, -1, 0, 0, -1, 0, 0, -1, 0, 0},
     {0, 0, -1, 0, 0, -1, 0, 0, -1, 0},
     {0, 0, 0, -1, 0, 0, -1, 0, 0, -1},
     {0, -1, 0, 0, -1, 0, 0, -1, 0, 0},
     {0, 0, -1, 0, 0, -1, 0, 0, -1, 0},
     {0, 0, 0, -1, 0, 0, -1, 0, 0, -1},
     {0, -1, 0, 0, -1, 0, 0, -1, 0, 0},
     {0, 0, -1, 0, 0, -1, 0, 0, -1, 0},
     {0, 0, 0, -1, 0, 0, -1, 0, 0, -1}},
    // pattern 2
    {{0, 0, 0, -1, 0, 0, -1, 0, 0, 0},
     {0, 0, -1, 0, 0, -1, 0, 0, -1, 0},
     {0, -1, 0, 0, -1, 0, 0, -1, 0, 0},
     {-1, 0, 0, -1, 0, 0, -1, 0, 0, 0},
     {0, 0, -1, 0, 0, -1, 0, 0, -1, 0},
     {0, -1, 0, 0, -1, 0, 0, -1, 0, 0},
     {-1, 0, 0, -1, 0, 0, -1, 0, 0, 0},
     {0, 0, -1, 0, 0, -1, 0, 0, -1, 0},
     {0, -1, 0, 0, -1, 0, 0, -1, 0, 0},
     {-1, 0, 0, -1, 0, 0, -1, 0, 0, 0}},
    // pattern 3
    {{0, 0, 0, -1, 0, 0, 0, -1, 0, 0},
     {0, -1, 0, -1, 0, -1, 0, -1, 0, -1},
     {0, -1, 0, -1, 0, -1, 0, -1, 0, -1},
     {0, -1, 0, -1, 0, -1, 0, -1, 0, -1},
     {0, -1, 0, -1, 0, -1, 0, -1, 0, -1},
     {0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
     {0, -1, 0, -1, 0, -1, 0, -1, 0, -1},
     {0, -1, 0, -1, 0, -1, 0, -1, 0, -1},
     {0, -1, 0, -1, 0, -1, 0, -1, 0, -1},
     {0, -1, 0, 0, 0, -1, 0, 0, 0, -1}},
    // pattern 4
    {{0, -1, -1, -1, -1, 0, -1, -1, -1, -1},
     {0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
     {-1, -1, -1, -1, -1, 0, -1, -1, -1, 0},
     {0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
     {0, -1, -1, -1, -1, 0, -1, -1, -1, -1},
     {0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
     {-1, -1, -1, -1, -1, 0, -1, -1, -1, 0},
     {0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
     {0, -1, -1, -1, -1, 0, -1, -1, -1, -1},
     {0, 0, 0, 0, 0, 0, 0, 0, 0, 0}},
};

class Environment {
public:
  // Environment(int rows, int cols) : rows(rows), cols(cols) {
  //   map.resize(rows, std::vector<int>(cols, 0));
  //   createWithPatterns();
  // }
  std::pair<int, int> robotPos;
  std::pair<int, int> packagePos;

  Environment(int rows, int cols) : rows(rows), cols(cols) {
    map.resize(rows, std::vector<int8_t>(cols, 0));
    nodeGrid.resize(rows, std::vector<Node>(cols));
    createWithPatterns();
    initializeNodes();
  }

  void createWithPatterns() {
    for (int i = 0; i < rows; i += patternSize) {
      for (int j = 0; j < cols; j += patternSize) {
        // Choose a pattern and place it in the map
        int patternIndex = std::rand() % numPatterns; // Random pattern index
        for (int pi = 0; pi < patternSize && i + pi < rows; ++pi) {
          for (int pj = 0; pj < patternSize && j + pj < cols; ++pj) {
            map[i + pi][j + pj] = patterns[patternIndex][pi][pj];
          }
        }
      }
    }
  }

  void saveToFile(const std::string &filename) {
    std::ofstream file(filename);
    if (!file.is_open()) {
      std::cerr << "Failed to open file for writing." << std::endl;
      return;
    }

    for (const auto &row : map) {
      for (size_t j = 0; j < row.size(); ++j) {
        file << row[j];
        if (j < row.size() - 1)
          file << ",";
      }
      file << "\n";
    }
  }

  void positionRobotAndPackage() {
    std::vector<std::pair<int, int>> freeSpaces;
    for (int i = 0; i < rows; ++i) {
      for (int j = 0; j < cols; ++j) {
        if (map[i][j] == 0) {
          freeSpaces.emplace_back(i, j);
        }
      }
    }

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> distrib(0, freeSpaces.size() - 1);

    // Randomly select two different positions for the robot and the package
    int index1 = distrib(gen);
    int index2 = distrib(gen);
    while (index1 == index2) {
      index2 = distrib(gen); // Ensure different positions
    }

    robotPos = freeSpaces[index1];
    packagePos = freeSpaces[index2];
  }

  void printRobotAndPackagePosition() {
    std::cout << "Robot Position: (" << robotPos.first << ", "
              << robotPos.second << ")\n";
    std::cout << "Package Position: (" << packagePos.first << ", "
              << packagePos.second << ")\n";
  }

  void initializeNodes() {
    for (int i = 0; i < rows; i++) {
      for (int j = 0; j < cols; j++) {
        nodeGrid[i][j].x = i;
        nodeGrid[i][j].y = j;
      }
    }
  }

private:
  int rows, cols;
  std::vector<std::vector<Node>> nodeGrid;
  std::vector<std::vector<int8_t>> map;
};

int main() {
  std::srand(static_cast<unsigned int>(time(nullptr)));

  Environment env(500, 500);
  env.saveToFile("../../wharehouse_ex/500_500.csv");

  env.positionRobotAndPackage();
  env.printRobotAndPackagePosition();

  int rows = 500;
  int cols = 500;
  size_t size = rows * cols * sizeof(Node);

  Node *d_nodes;
  hipMalloc(&d_nodes, size);

  // Assuming a block size of 16x16, adjust as necessary
  dim3 dimBlock(16, 16);
  dim3 dimGrid((cols + dimBlock.x - 1) / dimBlock.x,
               (rows + dimBlock.y - 1) / dimBlock.y);

  initializeNodesKernel<<<dimGrid, dimBlock>>>(d_nodes, rows, cols);

  hipDeviceSynchronize();

  int targetX = env.packagePos.first;
  int targetY = env.packagePos.second;

  aStarKernel<<<dimGrid, dimBlock>>>(d_nodes, rows, cols, targetX, targetY);
  hipDeviceSynchronize();

  hipFree(d_nodes);

  return 0;
}
